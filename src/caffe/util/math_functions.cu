#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#define TILE_WIDTH 16
#define BLOCK_SIZE 16 
namespace caffe {

  //gemv-------------------------------------------------------------------------------------------------------------------------------------------
  template<typename T>
  __global__ void matvec_kernel_ILP2(const T * __restrict__ dA, const T * __restrict__ dx, T * __restrict__ dy, const unsigned int nRows, const unsigned int nCols, const T alpha, const bool trans)
  {
    // LOG(INFO) << ("Warm hug from my kernle. ln 19.\n");
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  
    // printf("%d %d %d %d\n",  tid, threadIdx.x, blockIdx.x, blockDim.x);
    // printf("sdfasdf %d\n", gridDim.x * BLOCK_SIZE);
    // printf("%d\t%d\t%d\t%d\t\n", blockIdx.x, blockIdx.y,threadIdx.x,threadIdx.y);
  
    __shared__ T x_shared[BLOCK_SIZE];
  
    T y_val1 = 0.0;
    T y_val2 = 0.0;
  
    int tmp;
    if (trans) {tmp = ((nRows + BLOCK_SIZE - 1)/ BLOCK_SIZE);}
    else {tmp = ((nCols + BLOCK_SIZE - 1)/ BLOCK_SIZE);  }
  
    #pragma unroll
    for (unsigned int m = 0; m < tmp; ++m)
    {
      if (trans){
        if ((m * BLOCK_SIZE + threadIdx.x) <  nRows) 
          x_shared[threadIdx.x] = dx[threadIdx.x + m * BLOCK_SIZE];
        else
          x_shared[threadIdx.x] = 0.f;
      }
      else{
        if ((m * BLOCK_SIZE + threadIdx.x) <  nCols) 
          x_shared[threadIdx.x] = dx[threadIdx.x + m * BLOCK_SIZE];
        else
          x_shared[threadIdx.x] = 0.f;
      }
      
      __syncthreads();
  
      #pragma unroll
      for (unsigned int e = 0; e < BLOCK_SIZE; ++e) {
        if (trans){
          if ((tid) * nRows + (e + BLOCK_SIZE * m) < nRows*nCols){
            y_val1 += dA[(tid) * nRows + (e + BLOCK_SIZE * m)] * x_shared[e];
          }
          if ((tid + gridDim.x * BLOCK_SIZE) * nRows + (e + BLOCK_SIZE * m) < nRows*nCols){
            y_val2 += dA[(tid + gridDim.x * BLOCK_SIZE) * nRows + (e + BLOCK_SIZE * m)] * x_shared[e];
          }
        }
        else{
          if (tid + (e + BLOCK_SIZE * m) * nRows < nRows*nCols){
            y_val1 += dA[tid + (e + BLOCK_SIZE * m) * nRows] * x_shared[e];
          }
          if (tid + gridDim.x * BLOCK_SIZE + (e + BLOCK_SIZE * m) * nRows < nRows*nCols){
            y_val2 += dA[tid + gridDim.x * BLOCK_SIZE + (e + BLOCK_SIZE * m) * nRows] * x_shared[e];
          } 
        }
      }
  
      __syncthreads();
    }
    if (trans){
      if (tid < nCols) dy[tid] = y_val1 * alpha;
      if ((tid + gridDim.x * BLOCK_SIZE) < nCols) dy[tid + gridDim.x * BLOCK_SIZE] = y_val2 * alpha;
    }
    else{
      if (tid < nRows) dy[tid] = y_val1 * alpha;
      if ((tid + gridDim.x * BLOCK_SIZE) < nRows) dy[tid + gridDim.x * BLOCK_SIZE] = y_val2 * alpha;
    }
  }
  
  template <>
  void kk_gpu_gemv<float>(const float* h_A, const float* h_x, float* h_y, const unsigned int nRows, const unsigned int nCols, const float alpha, const bool trans) {
    int size = sizeof(float);
    // LOG(INFO) << ("Warm hug from my func. ln 51.\n");
    // LOG(INFO) << ("My caffe_gpu_gemv invoked in math_functions.cu.\n");
    float *d_A;
    float *d_x;
    float *d_y;
    if (trans){
      hipMalloc((void**)&d_A, nRows*nCols*size);
      hipMalloc((void**)&d_x, nRows*size);
      hipMalloc((void**)&d_y, nCols*size);
      hipMemcpy(d_A, h_A, nRows*nCols*size, hipMemcpyHostToDevice);
      hipMemcpy(d_x, h_x, nRows*size, hipMemcpyHostToDevice);
      hipMemcpy(d_y, h_y, nCols*size, hipMemcpyHostToDevice);
    }
    else{
      hipMalloc((void**)&d_A, nRows*nCols*size);
      hipMalloc((void**)&d_x, nCols*size);
      hipMalloc((void**)&d_y, nRows*size);
      hipMemcpy(d_A, h_A, nRows*nCols*size, hipMemcpyHostToDevice);
      hipMemcpy(d_x, h_x, nCols*size, hipMemcpyHostToDevice);
      hipMemcpy(d_y, h_y, nRows*size, hipMemcpyHostToDevice);
    }
    
    if (trans){
      dim3 dim_grid(((nCols+1)/2 + BLOCK_SIZE -1)/ BLOCK_SIZE);
      // printf("grid size %d\n", (nRows/2 + BLOCK_SIZE -1)/ BLOCK_SIZE);
      dim3 dim_block(BLOCK_SIZE);
      matvec_kernel_ILP2<float> <<<dim_grid, dim_block>>>(d_A, d_x, d_y, nRows, nCols, alpha, trans);
      hipMemcpy(h_y, d_y, nCols*size, hipMemcpyDeviceToHost);
    }
    else{
      dim3 dim_grid(((nRows+1)/2 + BLOCK_SIZE -1)/ BLOCK_SIZE);
      // printf("grid size %d\n", (nRows/2 + BLOCK_SIZE -1)/ BLOCK_SIZE);
      dim3 dim_block(BLOCK_SIZE); 
      matvec_kernel_ILP2<float> <<<dim_grid, dim_block>>>(d_A, d_x, d_y, nRows, nCols, alpha, trans);
      hipMemcpy(h_y, d_y, nRows*size, hipMemcpyDeviceToHost);
    }
    // printf("block_size %d\n", BLOCK_SIZE);
    // if (trans){
    //   matvec_kernel_ILP2<float> <<<dim_grid, dim_block>>>(d_A, d_x, d_y, nCols, nRows, alpha);
    //   hipMemcpy(h_y, d_y, nCols*size, hipMemcpyDeviceToHost);
    // }
    // else{
    // }
  
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
  }
  
  template <>
  void kk_gpu_gemv<double>(const double* h_A, const double* h_x, double* h_y, const unsigned int nRows, const unsigned int nCols, const double alpha, const bool trans) {
    int size = sizeof(double);
    // LOG(INFO) << ("Warm hug from my func. ln 51.\n");
    // LOG(INFO) << ("My caffe_gpu_gemv invoked in math_functions.cu.\n");
    double *d_A;
    double *d_x;
    double *d_y;
    if (trans){
      hipMalloc((void**)&d_A, nRows*nCols*size);
      hipMalloc((void**)&d_x, nRows*size);
      hipMalloc((void**)&d_y, nCols*size);
      hipMemcpy(d_A, h_A, nRows*nCols*size, hipMemcpyHostToDevice);
      hipMemcpy(d_x, h_x, nRows*size, hipMemcpyHostToDevice);
      hipMemcpy(d_y, h_y, nCols*size, hipMemcpyHostToDevice);
    }
    else{
      hipMalloc((void**)&d_A, nRows*nCols*size);
      hipMalloc((void**)&d_x, nCols*size);
      hipMalloc((void**)&d_y, nRows*size);
      hipMemcpy(d_A, h_A, nRows*nCols*size, hipMemcpyHostToDevice);
      hipMemcpy(d_x, h_x, nCols*size, hipMemcpyHostToDevice);
      hipMemcpy(d_y, h_y, nRows*size, hipMemcpyHostToDevice);
    }
    
    if (trans){
      dim3 dim_grid(((nCols+1)/2 + BLOCK_SIZE -1)/ BLOCK_SIZE);
      // printf("grid size %d\n", (nRows/2 + BLOCK_SIZE -1)/ BLOCK_SIZE);
      dim3 dim_block(BLOCK_SIZE);
      matvec_kernel_ILP2<double> <<<dim_grid, dim_block>>>(d_A, d_x, d_y, nRows, nCols, alpha, trans);
      hipMemcpy(h_y, d_y, nCols*size, hipMemcpyDeviceToHost);
    }
    else{
      dim3 dim_grid(((nRows+1)/2 + BLOCK_SIZE -1)/ BLOCK_SIZE);
      // printf("grid size %d\n", (nRows/2 + BLOCK_SIZE -1)/ BLOCK_SIZE);
      dim3 dim_block(BLOCK_SIZE); 
      matvec_kernel_ILP2<double> <<<dim_grid, dim_block>>>(d_A, d_x, d_y, nRows, nCols, alpha, trans);
      hipMemcpy(h_y, d_y, nRows*size, hipMemcpyDeviceToHost);
    }
    // printf("block_size %d\n", BLOCK_SIZE);
    // if (trans){
    //   matvec_kernel_ILP2<float> <<<dim_grid, dim_block>>>(d_A, d_x, d_y, nCols, nRows, alpha);
    //   hipMemcpy(h_y, d_y, nCols*size, hipMemcpyDeviceToHost);
    // }
    // else{
    // }
  
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
  }
  //gemv-------------------------------------------------------------------------------------------------------------------------------------------
  
//axpby
template <typename T>
__global__ void axpby_kernel(const int N, const T alpha, const T beta, const T*  X, T*  Y){
    CUDA_KERNEL_LOOP(i, N){
      // if (i < N)
        Y[i] = X[i]*alpha + Y[i]*beta;
    }
}

template <>
void kk_gpu_axpby<float>(const int N, const float alpha, const float beta, const float* X, float* Y){
    float *d_X, *d_Y;
    size_t size = N*sizeof(float);
    hipMalloc((void**)&d_X, size);
    hipMalloc((void**)&d_Y, size);
    hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice);
    dim3 dim_grid((N+BLOCK_SIZE)/BLOCK_SIZE);
    dim3 dim_block(BLOCK_SIZE);
    axpby_kernel<float> <<<dim_grid, dim_block>>>(N, alpha, beta, d_X, d_Y);
    hipMemcpy(Y, d_Y, size, hipMemcpyDeviceToHost);
    hipFree(d_X);
    hipFree(d_Y);
}
template <>
void kk_gpu_axpby<double>(const int N, const double alpha, const double beta, const double* X, double* Y){
  double *d_X, *d_Y;
    size_t size = N*sizeof(double);
    hipMalloc((void**)&d_X, size);
    hipMalloc((void**)&d_Y, size);
    hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice);
    dim3 dim_grid((N+BLOCK_SIZE)/BLOCK_SIZE);
    dim3 dim_block(BLOCK_SIZE);
    axpby_kernel<double> <<<dim_grid, dim_block>>>(N, alpha, beta, d_X, d_Y);
    hipMemcpy(Y, d_Y, size, hipMemcpyDeviceToHost);
    hipFree(d_X);
    hipFree(d_Y);
}
//axpby

//scale
template <typename T>
__global__ void scale_kernel(const int N, const T alpha, const T*  X, T* Y){
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    CUDA_KERNEL_LOOP(i, N){
      // if (i < N)
        Y[i] = X[i]*alpha;
    }
}

template <>
void kk_gpu_scale<float>(const int N, const float alpha, const float* X, float* Y){
    float *d_X, *d_Y;
    size_t size = N*sizeof(float);
    hipMalloc((void**)&d_X, size);
    hipMalloc((void**)&d_Y, size);
    hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);
    dim3 dim_grid((N+BLOCK_SIZE)/BLOCK_SIZE);
    dim3 dim_block(BLOCK_SIZE);
    scale_kernel<float> <<<dim_grid, dim_block>>>(N, alpha, d_X, d_Y);
    hipMemcpy(Y, d_Y, size, hipMemcpyDeviceToHost);
    hipFree(d_X);
    hipFree(d_Y);
}
template <>
void kk_gpu_scale<double>(const int N, const double alpha, const double* X, double* Y){
    double *d_X, *d_Y;
    size_t size = N*sizeof(double);
    hipMalloc((void**)&d_X, size);
    hipMalloc((void**)&d_Y, size);
    hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);
    dim3 dim_grid((N+BLOCK_SIZE)/BLOCK_SIZE);
    dim3 dim_block(BLOCK_SIZE);
    scale_kernel<double> <<<dim_grid, dim_block>>>(N, alpha, d_X, d_Y);
    hipMemcpy(Y, d_Y, size, hipMemcpyDeviceToHost);
    hipFree(d_X);
    hipFree(d_Y);
}
//scale

//axpy
template <typename T>
__global__ void axpy_kernel(const int N, const T alpha, const T*  X, T*  Y){
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    CUDA_KERNEL_LOOP(i, N){
      // if (i < N)
        Y[i] = X[i]*alpha + Y[i];
    }
}

template <>
void kk_gpu_axpy<float>(const int N, const float alpha, const float* X, float* Y){
    float *d_X, *d_Y;
    size_t size = N*sizeof(float);
    hipMalloc((void**)&d_X, size);
    hipMalloc((void**)&d_Y, size);
    hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice);
    dim3 dim_grid((N+BLOCK_SIZE)/BLOCK_SIZE);
    dim3 dim_block(BLOCK_SIZE);
    axpy_kernel<float> <<<dim_grid, dim_block>>>(N, alpha, d_X, d_Y);
    hipMemcpy(Y, d_Y, size, hipMemcpyDeviceToHost);
    hipFree(d_X);
    hipFree(d_Y);
}
template <>
void kk_gpu_axpy<double>(const int N, const double alpha, const double* X, double* Y){
    double *d_X, *d_Y;
    size_t size = N*sizeof(double);
    hipMalloc((void**)&d_X, size);
    hipMalloc((void**)&d_Y, size);
    hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice);
    dim3 dim_grid((N+BLOCK_SIZE)/BLOCK_SIZE);
    dim3 dim_block(BLOCK_SIZE);
    axpy_kernel<double> <<<dim_grid, dim_block>>>(N, alpha, d_X, d_Y);
    hipMemcpy(Y, d_Y, size, hipMemcpyDeviceToHost);
    hipFree(d_X);
    hipFree(d_Y);
}
//axpy

//dot
template <typename T>
__global__ void dot_kernel(const int N, const T* X, const T* Y, T* res){
    CUDA_KERNEL_LOOP(i, N){
      // if (i < N)
        res[i] = X[i]*Y[i];
    }
}

template <>
void kk_gpu_dot<float>(const int N, const float* X, const float* Y, float* res){
    float *d_X, *d_Y, *d_res;
    size_t size = N*sizeof(float);
    hipMalloc((void**)&d_X, size);
    hipMalloc((void**)&d_Y, size);
    hipMalloc((void**)&d_res, size);
    hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_res, res, size, hipMemcpyHostToDevice);
    dim3 dim_grid((N+BLOCK_SIZE)/BLOCK_SIZE);
    dim3 dim_block(BLOCK_SIZE);
    dot_kernel<float> <<<dim_grid, dim_block>>>(N, d_X, d_Y, d_res);
    hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_res);
}
template <>
void kk_gpu_dot<double>(const int N, const double* X, const double* Y, double* res){
  double *d_X, *d_Y, *d_res;
    size_t size = N*sizeof(double);
    hipMalloc((void**)&d_X, size);
    hipMalloc((void**)&d_Y, size);
    hipMalloc((void**)&d_res, size);
    hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_res, res, size, hipMemcpyHostToDevice);
    dim3 dim_grid((N+BLOCK_SIZE)/BLOCK_SIZE);
    dim3 dim_block(BLOCK_SIZE);
    dot_kernel<double> <<<dim_grid, dim_block>>>(N, d_X, d_Y, d_res);
    hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_res);
}
//dot

















using namespace std;
template<typename Dtype>
__global__ void MatrixMulKernle(int m, int n, int k, Dtype *A,Dtype  *B, Dtype *C,bool IsAdd_C,bool TransA,bool TransB)
{
    //申请共享内存，存在于每个block中
  __shared__ Dtype ds_A[TILE_WIDTH][TILE_WIDTH]; 
  __shared__ Dtype ds_B[TILE_WIDTH][TILE_WIDTH];
  
  //简化坐标记法,出现下面6个表示的地方就是并行的地方。
  int bx = blockIdx.x;		int by = blockIdx.y;
  int tx = threadIdx.x;		int ty = threadIdx.y;
  
  //确定结果矩阵中的行和列
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  
  //临时变量
  Dtype Cvalue=0;

  
  
  //循环读入A,B瓦片，计算结果矩阵，分阶段进行计算
  for (int t=0; t<(n-1)/TILE_WIDTH+1; ++t)
  {
    //将A,B矩阵瓦片化的结果放入shared memory中，每个线程加载相应于C元素的A/B矩阵元素
    if (Row < m && t * TILE_WIDTH + tx < n)		//越界处理，满足任意大小的矩阵相乘（可选）
      //ds_A[tx][ty] = A[t*TILE_WIDTH + tx][Row];
      if (TransA==false){
        ds_A[ty][tx] = A[Row*n+t*TILE_WIDTH+tx];//以合并的方式加载瓦片
        // printf("no transA");
      }
      else{
        // printf("uiou");
        ds_A[ty][tx] = A[(t*TILE_WIDTH+tx)*m+Row];//trans A[Row*n+t*TILE_WIDTH+tx];//以合并的方式加载瓦片,tx列，ty行

      }
        
    else
      ds_A[ty][tx] = 0.0;
  
    if (t * TILE_WIDTH + ty < n && Col < k)
      //ds_B[tx][ty] = B[Col][t*TILE_WIDTH + ty];
      if (TransB==false){
        ds_B[ty][tx] = B[(t*TILE_WIDTH + ty)*k+Col];

      }
      else{
        // printf("B");
        // ds_B[ty][tx] = B[(t*TILE_WIDTH + ty)*k+Col];
        // printf("%f ",ds_B[ty][tx]);
        ds_B[ty][tx] = B[(t*TILE_WIDTH + ty)+Col*n];
        // printf("%f\n",ds_B[ty][tx]);
       

      }
    else
      ds_B[ty][tx] = 0.0;	
  
    //保证tile中所有的元素被加载
    __syncthreads();
    
    if (IsAdd_C==0){
      Cvalue+=(ds_A[ty][0]*ds_B[0][tx]+ds_A[ty][1]*ds_B[1][tx]+ds_A[ty][2]*ds_B[2][tx]+ds_A[ty][3]*ds_B[3][tx]+ \
        ds_A[ty][4]*ds_B[4][tx]+ds_A[ty][5]*ds_B[5][tx]+ds_A[ty][6]*ds_B[6][tx]+ds_A[ty][7]*ds_B[7][tx]+ \
        ds_A[ty][8]*ds_B[8][tx]+ds_A[ty][9]*ds_B[9][tx]+ds_A[ty][10]*ds_B[10][tx]+ds_A[ty][11]*ds_B[11][tx]+ \
        ds_A[ty][12]*ds_B[12][tx]+ds_A[ty][13]*ds_B[13][tx]+ds_A[ty][14]*ds_B[14][tx]+ds_A[ty][15]*ds_B[15][tx]);
    }
    else{
      C[k*Row+Col]+=(ds_A[ty][0]*ds_B[0][tx]+ds_A[ty][1]*ds_B[1][tx]+ds_A[ty][2]*ds_B[2][tx]+ds_A[ty][3]*ds_B[3][tx]+ \
        ds_A[ty][4]*ds_B[4][tx]+ds_A[ty][5]*ds_B[5][tx]+ds_A[ty][6]*ds_B[6][tx]+ds_A[ty][7]*ds_B[7][tx]+ \
        ds_A[ty][8]*ds_B[8][tx]+ds_A[ty][9]*ds_B[9][tx]+ds_A[ty][10]*ds_B[10][tx]+ds_A[ty][11]*ds_B[11][tx]+ \
        ds_A[ty][12]*ds_B[12][tx]+ds_A[ty][13]*ds_B[13][tx]+ds_A[ty][14]*ds_B[14][tx]+ds_A[ty][15]*ds_B[15][tx]);
    }

    __syncthreads();
    if (IsAdd_C==0){
      if(Row < m && Col < k){
        // C[Col*m+Row]=Cvalue;
        C[k*Row+Col]=Cvalue;

      }
    }

    // for (int i = 0; i < TILE_WIDTH; ++i){
    //   if (IsAdd_C==0){
    //     Cvalue += ds_A[ty][i] * ds_B[i][tx];//从shared memory中取值

    //   }
    //   else{
    //     if(Row < m && Col < k)
    //       // C[Col*m+Row]+= ds_A[ty][i] * ds_B[i][tx];
    //       C[k*Row+Col]+= ds_A[ty][i] * ds_B[i][tx];

    //   }
    // }
    
  
    // //确保所有线程完成计算后，进行下一个阶段的计算
    // __syncthreads();
    // if (IsAdd_C==0){
    //   if(Row < m && Col < k){
    //     // C[Col*m+Row]=Cvalue;
    //     C[k*Row+Col]=Cvalue;

    //   }
    // }





    // if(Row < m && Col < k){
    //   if (IsAdd_C==0){
    //     C[Col*m+Row]=Cvalue;
    //   }
    //   else{
    //     printf("%d %d %f %f\n",Col,Row,C[Col*m+Row],Cvalue);
    //     C[Col*m+Row]=Cvalue;
    //     // printf("wyiyiyyyi\n");
    //   }
    // }

      

  }
}
  



template <>
void caffe_gpu_gemm<float>(const int m, const int n, const int k,
     const float* A, const float* B,   float* C,bool IsAdd_C,bool TransA,bool TransB) {
  // Note that cublas follows fortran order.
  // printf("2444444333333333\n");

  //lzy
  //分配显存空间
  int size = sizeof(float);
  float *d_a;
  float *d_b;
  float *d_c;
  hipMalloc((void**)&d_a,m*n*size);
  hipMalloc((void**)&d_b,n*k*size);
  hipMalloc((void**)&d_c,m*k*size);

  //把数据从Host传到Device
  hipMemcpy(d_a, A, size*m*n, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, size*n*k, hipMemcpyHostToDevice);
  hipMemcpy(d_c, C, size*m*k, hipMemcpyHostToDevice);


  //分配网格结构
  int tile_width=TILE_WIDTH;
  dim3 dimGrid((k-1)/tile_width+1,(m-1)/tile_width+1,1);	//向上取整
  dim3 dimBlock(tile_width,tile_width,1);
  //lzy
  MatrixMulKernle<float><<<dimGrid,dimBlock>>>(m,n,k,d_a,d_b,d_c,IsAdd_C,TransA,TransB);
  // MatrixMulKernle<float><<<dimGrid,dimBlock>>>(m,k,n,d_b,d_a,d_c,IsAdd_C,TransA,TransB);

  hipMemcpy(C, d_c, size*m*k, hipMemcpyDeviceToHost);

	hipFree(d_c);
  hipFree(d_a);
	hipFree(d_b);
}

template <>
void caffe_gpu_gemm<double>(const int m, const int n, const int k,
     const double* A, const double* B,   double* C,bool IsAdd_C,bool TransA,bool TransB) {
  // Note that cublas follows fortran order.
  // printf("24444433333333\n");

  //分配显存空间
  int size = sizeof(double);
  double *d_a;
  double *d_b;
  double *d_c;
  hipMalloc((void**)&d_a,m*n*size);
  hipMalloc((void**)&d_b,n*k*size);
  hipMalloc((void**)&d_c,m*k*size);

  //把数据从Host传到Device
  hipMemcpy(d_a, A, size*m*n, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, size*n*k, hipMemcpyHostToDevice);
  hipMemcpy(d_c, C, size*m*k, hipMemcpyHostToDevice);


  //分配网格结构
  int tile_width=TILE_WIDTH;
  dim3 dimGrid((k-1)/tile_width+1,(m-1)/tile_width+1,1);	//向上取整
  dim3 dimBlock(tile_width,tile_width,1);
  //lzy
  MatrixMulKernle<double><<<dimGrid,dimBlock>>>(m,n,k,d_a,d_b,d_c,IsAdd_C,TransA,TransB);
  // MatrixMulKernle<double><<<dimGrid,dimBlock>>>(k,n,m,d_b,d_a,d_c,IsAdd_C,TransA,TransB);
// 
  hipMemcpy(C, d_c, size*m*k, hipMemcpyDeviceToHost);

	hipFree(d_c);
  hipFree(d_a);
	hipFree(d_b);
}









template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  // printf("2333333333333333333\n");
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
      // LOG(INFO) << ("caffe_gpu_gemv invoked in math_functions.cu.\n");
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float* X,
                           hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double* X,
                            hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <typename Dtype>
__global__ void sqrt_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = sqrt(a[index]);
  }
}

template <>
void caffe_gpu_sqrt<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_sqrt<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

}  // namespace caffe
